#include "hip/hip_runtime.h"
#include <cstdio>
#include <thrust/scan.h>
#include <thrust/device_vector.h>

__global__ void computeLengths(int* data, int* lengths, int* symbolSizes, int n) {
    int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (thid >= n) return;
    lengths[thid] = symbolSizes[data[thid]];
    //printf("%d %d\n", thid, lengths[thid]);
}

int* toIntPtr(thrust::device_vector<int>& v) {
    return thrust::raw_pointer_cast(&v[0]);
}

__global__ void computeFirstByteSym(int* lengthSum, int* firstByteSym, int n, int outSize) {
    int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (thid >= n || thid == 0) return;

    int curr = (lengthSum[thid - 1] / 8);
    int prev = (thid == 1) ? 0 : (lengthSum[thid - 2]/8);
    if (prev != curr) {
        //printf("sym: %d, curr: %d\n", thid, curr);
        firstByteSym[curr] = thid;
    }
}

__global__ void computeResult(int* symbols, int* data, int* firstByteSym, int* lengthSum, int* results, int n, int outSize) {
    int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (thid >= outSize) return;

    int symI = firstByteSym[thid];
    if (symI != 0) symI --;
    int offsetDelta = thid * 8;

    //printf("thid: %d, sym: %d\n", thid, symI);
    int result = 0;
    while (symI < n) {
        int shift = (symI == 0 ? 0 : lengthSum[symI-1]) - offsetDelta;
        //printf("thid: %d | symI: %d, shift: %d\n", thid, symI, shift);
        if (shift > 8) break;
        int code = symbols[data[symI]];
        if (shift >= 0)
            result |= code << shift;
        else
            result |= code >> (-shift);
        symI ++;
    }
    results[thid] = result;
}

int main(){
    hipSetDevice(1);
    int symbolsData[] = {0b1, 0b01, 0b001, 0b000};
    int sizesData[] = {1, 2, 3, 3};
    const int symN = 4;
    const int dataN = 10000;

    thrust::device_vector<int> data;
    for (int i=0; i < dataN; i ++) data.push_back(i % symN);
    thrust::device_vector<int> symbols (symbolsData, symbolsData + symN);
    thrust::device_vector<int> symbolSizes (sizesData, sizesData + symN);

    thrust::device_vector<int> lengthSum (dataN);

    computeLengths<<<(dataN + 1023) / 1024, 1024>>>
        (toIntPtr(data), toIntPtr(lengthSum),
         toIntPtr(symbolSizes), dataN);
    thrust::inclusive_scan(lengthSum.begin(), lengthSum.end(), lengthSum.begin());

    //for (int i : lengthSum) printf("l=%d\n", i);

    int outSize = (lengthSum.back() / 8 + 1);

    thrust::device_vector<int> firstByteSym;
    firstByteSym.resize(outSize, 0);
    computeFirstByteSym<<<(outSize + 1023) / 1024, 1024>>>
        (toIntPtr(lengthSum),
         toIntPtr(firstByteSym),
         dataN, outSize);

    //for (int i : firstByteSym) printf("s=%d\n", i);

    thrust::device_vector<int> result;
    result.resize(outSize, 0);

    computeResult<<<(outSize + 1023) / 1024, 1024>>>
        (toIntPtr(symbols),
         toIntPtr(data),
         toIntPtr(firstByteSym),
         toIntPtr(lengthSum),
         toIntPtr(result),
         dataN, outSize);

    //for (int i : result) printf("result=%d\n", i);
    hipDeviceSynchronize ();
    return 0;
}
